#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""


#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <chrono>


#define FILENAME "vector_sequences_file.txt"
#define DEV "DEV_vector_sequences_file.txt"

#define MAX_WORD_SIZE 12000 // in int32_t size

__global__ void hamming_one(int32_t* d_data, int N, int P);
__device__ uint32_t count_set_bits(int32_t n);
__device__ int get_shm_starting_index(int id, int ints_per_word);

void generate_bit_sequences(const int N, const int L);
void read_bit_sequences(std::string path, int32_t*& h_input, int& N, int& L, int& ints_per_word);
int32_t set_bit(int32_t n, int k, int b);
void set_thread_and_block_counts(int N, int ints_per_word, int& block_count, int& thread_count);
void initialize_CUDA_memory(int N, int P);
void clean();

void print_time_difference(std::chrono::steady_clock::time_point start, std::chrono::steady_clock::time_point end);


const int BITS_IN_INT = (8 * sizeof(int32_t));
int32_t* h_data, * d_data;


int main()
{

    int N, L, ints_per_word;

    generate_bit_sequences(100000, 1000);

    read_bit_sequences(DEV, h_data, N, L, ints_per_word);
    
    initialize_CUDA_memory(N, ints_per_word);

    int block_count, thread_count;
    set_thread_and_block_counts(N, ints_per_word, block_count, thread_count);

    std::chrono::steady_clock::time_point start = std::chrono::high_resolution_clock::now();
    std::cout << "Comparing... \n";
    hamming_one <<<block_count, thread_count >>> (d_data, N, ints_per_word);
    hipDeviceSynchronize();
    std::chrono::steady_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::cout << "End of comparing... ";
    print_time_difference(start, end);

    clean();
    std::cout << "End of Program\n";
    return 0;
}


__global__ void hamming_one(int32_t* d_data, int N, int ints_per_word)
{
    __shared__ int32_t shm[MAX_WORD_SIZE];
    int key_word_id = blockIdx.x * blockDim.x + threadIdx.x;
    int hamming_distance;
    int shm_id = get_shm_starting_index(threadIdx.x, ints_per_word);

    for (int i = 0; i < ints_per_word; i++)
    {
        // put thread's key word into shared memory
        shm[shm_id + i * 32] = d_data[key_word_id * ints_per_word + i];
    }

    // compare each word that is after key word
    for (int compare_word_id = key_word_id + 1; compare_word_id < N; compare_word_id++)
    {
        hamming_distance = 0;

        // compare each part of word
        for (int i = 0; i < ints_per_word; i++)
        {
            int32_t xor =shm[shm_id + i * 32] ^ d_data[compare_word_id * ints_per_word + i];
            hamming_distance += count_set_bits(xor);
            if (hamming_distance > 1) break; // if hamming distance is greater than 1 stop checking
        }

        if (hamming_distance == 1) printf("Pair [%d] and [%d]\n", key_word_id, compare_word_id);

    }

}

__device__ uint32_t count_set_bits(int32_t n)
{
    if (n == 0) return 0; // all zeros
    if ((n & (n - 1)) == 0) return 1; // is power of 2
    return 2; // more  than one 1
}

__device__ int get_shm_starting_index(int id, int ints_per_word)
{
    int id_32 = id / 32;
    int id_bank = id % 32;
    return id_32 * (32 * ints_per_word) + id_bank; // offset + id
}

void generate_bit_sequences(const int N, const int L)
{
    std::cout << "Begin generation ...\n";
    std::chrono::steady_clock::time_point start = std::chrono::high_resolution_clock::now();
    srand((unsigned)time(NULL));
    std::ofstream out(FILENAME);
    out << N << std::endl << L << std::endl;
    int number;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < L; j++)
        {
            number = rand() % 2;
            number == 0 ? out << "0" : out << "1";
        }
        out << std::endl;
    }
    std::chrono::steady_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::cout << "End generation ... ";
    print_time_difference(start, end);
    out.close();
}

void  read_bit_sequences(std::string path, int32_t*& h_input, int& N, int& L, int& ints_per_word)
{
    std::ifstream stream;
    std::chrono::steady_clock::time_point start = std::chrono::high_resolution_clock::now();
    try
    {
        std::cout << "Started reading file\n";
        stream.open(path, std::ios::in);
        stream >> N >> L;

        ints_per_word = ceil((double)L / BITS_IN_INT);
        
        h_input = new int32_t[N * ints_per_word];
        int curr_int = 0;
        int curr_bit = BITS_IN_INT - 1;

        for (int i = 0; i < N * ints_per_word; ++i)
            h_input[i] = 0;

        for (int i = 0; i < N; ++i)
        {
            for (int j = 0; j < L; ++j)
            {
                char ch;
                stream >> ch;
                h_input[curr_int] = set_bit(h_input[curr_int], curr_bit, ch == '1' ? 1 : 0);
                curr_bit--;
                if (curr_bit == -1)
                {
                    curr_bit = BITS_IN_INT - 1;
                    curr_int++;
                }
            }
            curr_int++;
            curr_bit = BITS_IN_INT - 1;
        }

        stream.close();
        std::chrono::steady_clock::time_point end = std::chrono::high_resolution_clock::now();
        std::cout << "Successfully read file ";
        print_time_difference(start, end);
    }
    catch (...)
    {
        std::chrono::steady_clock::time_point end = std::chrono::high_resolution_clock::now();
        std::cout << "Error handling input file ";
        print_time_difference(start, end);
        return;
    }
}

int32_t set_bit(int32_t n, int k, int b)
{
    // set k-th bit of n to bit b
    int32_t mask = 1 << k;
    return ((n & ~mask) | (b << k));
}

void set_thread_and_block_counts(int N, int ints_per_word, int& block_count, int& thread_count)
{
    thread_count = MAX_WORD_SIZE / ints_per_word - 32;
    block_count = ceil((double)N / thread_count);
}

void initialize_CUDA_memory(int N, int ints_per_word)
{
    // allocate memory
    hipMalloc(&d_data, N * ints_per_word * sizeof(int32_t));
    // copy data from host  to device
    hipMemcpy(d_data, h_data, N * ints_per_word * sizeof(int32_t), hipMemcpyHostToDevice);
    std::cout << "CUDA memory initialized\n";
}

void clean()
{
    std::cout << "Clean\n";
    hipFree(d_data);
    delete[] h_data;
}


void print_time_difference(std::chrono::steady_clock::time_point start, std::chrono::steady_clock::time_point end)
{
    double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_taken *= 1e-9;
    printf("[It took %.9f sec]\n", time_taken);
    
}